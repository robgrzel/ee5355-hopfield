#include "hip/hip_runtime.h"

#include "hopfield.hpp"

#include <cstdint>
#include <cassert>
#include <vector>
#include <iostream>
using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
__global__ void gpu_sparse_ell_coal_recall_kernel(size_t size,
                                        bool * state,
                                        float * thresholds,
                                        float * ell_w_nnz,
                                        int * ell_w_colInd,
                                        int max_elements ,
                                        bool * stable) 
{
  // TODO
   size_t node = blockIdx.x * blockDim.x + threadIdx.x;
   if (node < size) {
    float value = 0.0f;

    for (size_t k = node; k <max_elements*size ; k += size)
    {
        if (state[ell_w_colInd[k]])
                value += ell_w_nnz[k];
        else
                value -= ell_w_nnz[k];
    }

    bool update = value > thresholds[node];
    if (update != state[node]) {
      *stable = false;
      state[node] = update;
    }
  }
 
}

GPUSparseELLCoalHopfieldNetwork::GPUSparseELLCoalHopfieldNetwork(const std::vector<float> &thresholds,
                                                   const std::vector<std::vector<float>> &weights,
                                                   float weightThreshold) :
  SparseHopfieldNetwork(thresholds, weights, weightThreshold) {
  //Converting CSR to ELL
  CSR_2_ELL();
/*printf("\n   ELL   \n");
   for (int f=0; f<max_elements*w_row;++f)
      printf("%.2f  ",ell_w_nnz[f]);
   printf("\n");
   for (int f=0; f<max_elements*w_row;++f)
      printf("%d  ",ell_w_colInd[f]);
   printf("\n");
      printf("%d  ",max_elements);
   printf("\n");
*/
 
  //Allocating device memory
  gpuErrchk(hipMalloc((void**)&state_d,sizeof(bool) * size));
  gpuErrchk(hipMalloc((void**)&stable_d,sizeof(bool)));
  gpuErrchk(hipMalloc((void**)&threshold_d,sizeof(float) * size));
  gpuErrchk(hipMalloc((void**)&ell_w_nnz_d,sizeof(float) * (max_elements*w_row)));
  gpuErrchk(hipMalloc((void**)&ell_w_colInd_d,sizeof(int) * (max_elements*w_row)));
  

  // Copying data to device
  gpuErrchk(hipMemcpy(threshold_d, thresholds.data(), size * sizeof(float),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(ell_w_nnz_d, ell_w_nnzT.data(), max_elements*w_row*sizeof(float),hipMemcpyHostToDevice));
  gpuErrchk(hipMemcpy(ell_w_colInd_d, ell_w_colIndT.data(), max_elements*w_row*sizeof(int),hipMemcpyHostToDevice));


}

GPUSparseELLCoalHopfieldNetwork::~GPUSparseELLCoalHopfieldNetwork() {

  //Free Device memory
  hipFree(state_d);
  hipFree(threshold_d);
  hipFree(ell_w_nnz_d);
  hipFree(ell_w_colInd_d);
  hipFree(stable_d);

}

vector<bool> GPUSparseELLCoalHopfieldNetwork::evaluate(const vector<bool> &data) {

  bool stable_h;
  bool data_h[size];
  unsigned numThreads = 256;
  unsigned numBlocks = (size-1)/numThreads+1;

  copy(data.begin(), data.end(), data_h);

  gpuErrchk(hipMemcpy(state_d, data_h, size * sizeof(bool),hipMemcpyHostToDevice));
  do {
    stable_h = true;
    gpuErrchk(hipMemcpy(stable_d, &stable_h, sizeof(bool),
                         hipMemcpyHostToDevice));

    gpu_sparse_ell_coal_recall_kernel<<< numBlocks, numThreads >>> 
    (size, state_d, threshold_d, ell_w_nnz_d, ell_w_colInd_d, max_elements, stable_d);

    gpuErrchk(hipDeviceSynchronize());

  //gpuErrchk(hipMemcpy(data_h, state_d, size * sizeof(bool),hipMemcpyDeviceToHost)); for(int i=0; i< size; ++i) cout << data_h[i]; cout << endl;
    gpuErrchk(hipMemcpy(&stable_h, stable_d, sizeof(bool),
                         hipMemcpyDeviceToHost));
  } while (!stable_h);

  gpuErrchk(hipMemcpy(data_h, state_d, size * sizeof(bool),
                       hipMemcpyDeviceToHost));

  gpuErrchk(hipDeviceSynchronize());
  
  vector<bool> state(data_h, data_h + size);


  return state;

}
