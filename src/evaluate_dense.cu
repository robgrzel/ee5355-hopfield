#include "hip/hip_runtime.h"

#include "hopfield.hpp"

#include <cstdint>
#include <cassert>
#include <vector>
#include <iostream>
using namespace std;

__global__ void gpu_dense_recall_kernel(size_t size,
                                        bool * state,
                                        float * thresholds,
                                        float * weights,
                                        bool * stable) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  float value = 0.0f;
  bool update;

  bool stableT = true;
  
  printf("%d !!!!\n", i);

  if (i < size) {
    for (size_t k = 0; k < size; ++k) {
      if (state[k])
        value += weights[i * size + k];
      else
        value -= weights[i * size + k];
    }

    update = value > thresholds[i];
    printf("%d : %f\n", i, value);
    if (update != state[i]) {
      stableT = false;
      state[i] = update;
      printf("Updating %d to %d\n", i, update);
    }

    //TODO: use reduction to find stable
    atomicAnd((int *) stable, (int) stableT);
  }
}

GPUDenseHopfieldNetwork::GPUDenseHopfieldNetwork(const std::vector<float> &thresholds,
                                                 const std::vector<std::vector<float>> &weights) :
    HopfieldNetwork(thresholds, weights) {
  assert(hipMalloc((void**) &thresholdsDev, sizeof(float) * size) == hipSuccess);
  assert(hipMalloc((void**) &weights, sizeof(float) * size * size) == hipSuccess);

  float (*weightArray)[size] = (float(*)[size])new float[size * size];
  for (size_t i = 0; i < size; ++i) {
    for (size_t j = 0; j < size; ++j) {
      weightArray[i][j] = weights[i][j];
    }
  }

  assert(hipMemcpy(thresholdsDev, thresholds.data(), size * sizeof(float),
                    hipMemcpyHostToDevice) == hipSuccess);
  assert(hipMemcpy(weightsDev, weightArray, size * size * sizeof(float),
                    hipMemcpyHostToDevice) == hipSuccess);
  
  delete[] weightArray; 
}

GPUDenseHopfieldNetwork::~GPUDenseHopfieldNetwork() {
  hipFree(thresholdsDev);
  hipFree(weightsDev);
}

vector<bool> GPUDenseHopfieldNetwork::evaluate(const vector<bool> &data) {
  bool stable;
  bool dataArray[size];

  bool *stateDev;
  bool *stableDev;
  unsigned numThreads = 256;
  unsigned numBlocks = size / numThreads;

  if (size % numThreads) numBlocks++;

  assert(hipMalloc((void**) &stateDev, sizeof(bool) * size) == hipSuccess);
  assert(hipMalloc((void**) &stableDev, sizeof(bool)) == hipSuccess);

  assert(hipMemcpy(stateDev, data.data(), size * sizeof(bool),
                    hipMemcpyHostToDevice) == hipSuccess);

  do {
    stable = true;
    assert(hipMemcpy(stableDev, &stable, sizeof(bool),
                      hipMemcpyHostToDevice) == hipSuccess);

    gpu_dense_recall_kernel<<< numBlocks, numThreads >>>
      (size, stateDev, thresholdsDev, weightsDev, stableDev);
    assert(hipDeviceSynchronize() == hipSuccess);

    assert(hipMemcpy(&stable, stableDev, sizeof(bool),
                      hipMemcpyDeviceToHost) == hipSuccess);
  } while (!stable);

  assert(hipMemcpy(dataArray, stateDev, size * sizeof(bool),
                    hipMemcpyDeviceToHost) == hipSuccess);

  assert(hipDeviceSynchronize() == hipSuccess);
  
  vector<bool> state(dataArray, dataArray + size);
  /*for (size_t i = 0; i < size; ++i) {
    cout << state[i] << " ";
  }
  cout << endl;*/

  hipFree(stateDev);
  hipFree(stableDev);

  return state;
}

