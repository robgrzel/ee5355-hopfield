#include "hip/hip_runtime.h"

#include "hopfield.hpp"

#include <cstdint>
#include <cassert>
#include <vector>
#include <iostream>
using namespace std;

#define BLOCK_SIZE 32

__global__ void gpu_dense_recall_kernel(size_t size,
                                        bool * state,
                                        float * thresholds,
                                        float * weights,
                                        bool * stable) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < size) {
    float value = 0.0f;
    for (size_t k = 0; k < size; ++k) {
      if (state[k])
        value += weights[i * size + k];
      else
        value -= weights[i * size + k];
    }

    bool update = value > thresholds[i];
    if (update != state[i]) {
      *stable = false;
    }
    state[i] = update;
  }
}

GPUDenseHopfieldNetwork::GPUDenseHopfieldNetwork(const std::vector<float> &thresholds,
                                                 const std::vector<std::vector<float>> &weights) :
  HopfieldNetwork(thresholds, weights) {
  cudaCheck(hipMalloc((void**) &thresholdsDev, sizeof(float) * size));
  cudaCheck(hipMalloc((void**) &weightsDev, sizeof(float) * size * size));

  float (*weightArray)[size] = (float(*)[size])new float[size * size];
  for (size_t i = 0; i < size; ++i) {
    copy(weights[i].begin(), weights[i].end(), weightArray[i]);
  }

  cudaCheck(hipMemcpy(thresholdsDev, thresholds.data(), size * sizeof(float),
                       hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(weightsDev, weightArray, size * size * sizeof(float),
                       hipMemcpyHostToDevice));
  
  delete[] weightArray; 
}

GPUDenseHopfieldNetwork::~GPUDenseHopfieldNetwork() {
  hipFree(thresholdsDev);
  hipFree(weightsDev);
}

vector<bool> GPUDenseHopfieldNetwork::evaluate(const vector<bool> &data) {
  bool stable;
  bool dataArray[size];

  bool *stateDev;
  bool *stableDev;
  unsigned numBlocks = size / BLOCK_SIZE;

  if (size % BLOCK_SIZE) numBlocks++;

  cudaCheck(hipMalloc((void**) &stateDev, sizeof(bool) * size));
  cudaCheck(hipMalloc((void**) &stableDev, sizeof(bool)));

  copy(data.begin(), data.end(), dataArray);
  cudaCheck(hipMemcpy(stateDev, dataArray, size * sizeof(bool),
                       hipMemcpyHostToDevice));

  do {
    stable = true;
    cudaCheck(hipMemcpy(stableDev, &stable, sizeof(bool),
                         hipMemcpyHostToDevice));

    gpu_dense_recall_kernel<<< numBlocks, BLOCK_SIZE >>>
      (size, stateDev, thresholdsDev, weightsDev, stableDev);
    cudaCheck(hipDeviceSynchronize());

    cudaCheck(hipMemcpy(&stable, stableDev, sizeof(bool),
                         hipMemcpyDeviceToHost));
  } while (!stable);

  cudaCheck(hipMemcpy(dataArray, stateDev, size * sizeof(bool),
                       hipMemcpyDeviceToHost));

  cudaCheck(hipDeviceSynchronize());
  
  vector<bool> state(dataArray, dataArray + size);

  hipFree(stateDev);
  hipFree(stableDev);

  return state;
}

