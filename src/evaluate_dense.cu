#include "hip/hip_runtime.h"

#include "hopfield.hpp"

#include <cstdint>
#include <cassert>
#include <vector>
#include <iostream>
using namespace std;

__global__ void gpu_dense_recall_kernel(size_t size,
                                        bool * state,
                                        float * thresholds,
                                        float * weights,
                                        bool * stable) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  float value = 0.0f;
  bool update;

  bool stableT = true;

  if (i < size) {
    for (size_t k = 0; k < size; ++k) {
      if (state[k])
        value += weights[i * size + k];
      else
        value -= weights[i * size + k];
    }

    update = value > thresholds[i];
    if (update != state[i]) {
      stableT = false;
      state[i] = update;
    }

    //TODO: use reduction to find stable
    atomicAnd((int *) stable, (int) stableT);
  }
}

GPUDenseHopfieldNetwork::GPUDenseHopfieldNetwork(const std::vector<float> &thresholds,
                                                 const std::vector<std::vector<float>> &weights) :
  HopfieldNetwork(thresholds, weights) {
  cudaCheck(hipMalloc((void**) &thresholdsDev, sizeof(float) * size));
  cudaCheck(hipMalloc((void**) &weightsDev, sizeof(float) * size * size));

  float (*weightArray)[size] = (float(*)[size])new float[size * size];
  for (size_t i = 0; i < size; ++i) {
    for (size_t j = 0; j < size; ++j) {
      weightArray[i][j] = weights[i][j];
    }
  }

  cudaCheck(hipMemcpy(thresholdsDev, thresholds.data(), size * sizeof(float),
                       hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(weightsDev, weightArray, size * size * sizeof(float),
                       hipMemcpyHostToDevice));
  
  delete[] weightArray; 
}

GPUDenseHopfieldNetwork::~GPUDenseHopfieldNetwork() {
  hipFree(thresholdsDev);
  hipFree(weightsDev);
}

vector<bool> GPUDenseHopfieldNetwork::evaluate(const vector<bool> &data) {
  bool stable;
  bool dataArray[size];

  bool *stateDev;
  bool *stableDev;
  unsigned numThreads = 256;
  unsigned numBlocks = size / numThreads;

  if (size % numThreads) numBlocks++;

  cudaCheck(hipMalloc((void**) &stateDev, sizeof(bool) * size));
  cudaCheck(hipMalloc((void**) &stableDev, sizeof(bool)));

  copy(data.begin(), data.end(), dataArray);
  cudaCheck(hipMemcpy(stateDev, dataArray, size * sizeof(bool),
                       hipMemcpyHostToDevice));

  do {
    stable = true;
    cudaCheck(hipMemcpy(stableDev, &stable, sizeof(bool),
                         hipMemcpyHostToDevice));

    gpu_dense_recall_kernel<<< numBlocks, numThreads >>>
      (size, stateDev, thresholdsDev, weightsDev, stableDev);
    cudaCheck(hipDeviceSynchronize());

    cudaCheck(hipMemcpy(&stable, stableDev, sizeof(bool),
                         hipMemcpyDeviceToHost));
  } while (!stable);

  cudaCheck(hipMemcpy(dataArray, stateDev, size * sizeof(bool),
                       hipMemcpyDeviceToHost));

  cudaCheck(hipDeviceSynchronize());
  
  vector<bool> state(dataArray, dataArray + size);

  hipFree(stateDev);
  hipFree(stableDev);

  return state;
}

