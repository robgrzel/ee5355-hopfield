#include "hip/hip_runtime.h"

#include "hopfield.hpp"

#include <cstdint>
#include <cassert>
#include <vector>
#include <iostream>
using namespace std;

#define WORD uint32_t
#define WORD_SIZE 32
#define BLOCK_SIZE 256

__global__ void gpu_dense_bit_recall_kernel(size_t size,
                                            bool *state, //WORD *state,
                                            float *thresholds,
                                            float *weights,
                                            bool *stable) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < size) {
    float value = 0.0f;
    for (size_t k = 0; k < size; ++k) {
      if (state[k])
        value += weights[i * size + k];
      else
        value -= weights[i * size + k];
    }

    bool update = value > thresholds[i];
    if (update != state[i]) {
      *stable = false;
      state[i] = update;
    }
  }
}

GPUDenseBitHopfieldNetwork::GPUDenseBitHopfieldNetwork(const std::vector<float> &thresholds,
                                                       const std::vector<std::vector<float>> &weights) :
  HopfieldNetwork(thresholds, weights) {
  cudaCheck(hipMalloc((void**) &thresholdsDev, sizeof(float) * size));
  cudaCheck(hipMalloc((void**) &weightsDev, sizeof(float) * size * size));

  float (*weightArray)[size] = (float(*)[size])new float[size * size];
  for (size_t i = 0; i < size; ++i) {
    copy(weights[i].begin(), weights[i].end(), weightArray[i]);
  }

  cudaCheck(hipMemcpy(thresholdsDev, thresholds.data(), size * sizeof(float),
                       hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(weightsDev, weightArray, size * size * sizeof(float),
                       hipMemcpyHostToDevice));
  
                       delete[] weightArray;
}

GPUDenseBitHopfieldNetwork::~GPUDenseBitHopfieldNetwork() {
  hipFree(thresholdsDev);
  hipFree(weightsDev);
}

vector<bool> GPUDenseBitHopfieldNetwork::evaluate(const vector<bool> &data) {
  bool stable;
  bool dataArray[size];

  bool *stateDev;
  bool *stableDev;
  
  unsigned numThreads = 256;
  unsigned numBlocks = size / numThreads;

  if (size % numThreads) numBlocks++;

  cudaCheck(hipMalloc((void**) &stateDev, sizeof(bool) * size));
  cudaCheck(hipMalloc((void**) &stableDev, sizeof(bool)));

  copy(data.begin(), data.end(), dataArray);
  cudaCheck(hipMemcpy(stateDev, dataArray, size * sizeof(bool),
                       hipMemcpyHostToDevice));

  do {
    stable = true;
    cudaCheck(hipMemcpy(stableDev, &stable, sizeof(bool),
                         hipMemcpyHostToDevice));

    gpu_dense_bit_recall_kernel<<< numBlocks, numThreads >>>
      (size, stateDev, thresholdsDev, weightsDev, stableDev);
    cudaCheck(hipDeviceSynchronize());

    cudaCheck(hipMemcpy(&stable, stableDev, sizeof(bool),
                         hipMemcpyDeviceToHost));
  } while (!stable);

  cudaCheck(hipMemcpy(dataArray, stateDev, size * sizeof(bool),
                       hipMemcpyDeviceToHost));

  cudaCheck(hipDeviceSynchronize());
  
  vector<bool> state(dataArray, dataArray + size);

  hipFree(stateDev);
  hipFree(stableDev);

  return state;
}

