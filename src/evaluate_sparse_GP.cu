#include "hip/hip_runtime.h"

#include "hopfield.hpp"

#include <cstdint>
#include <cassert>
#include <vector>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <time.h>
using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}





__global__ void gpu_sparse_gpu_prePro_recall_kernel(size_t size,
                                        bool * state,
                                        float * thresholds,
                                        float * sW_nnz,
                                        int * sW_colInd,
                                        int * sW_rowPtr,
                                        bool * stable) 
{
  // TODO
   size_t node = blockIdx.x * blockDim.x + threadIdx.x;
   if (node < size) {
    float value = 0.0f;
    for (size_t k = sW_rowPtr[node]; k < sW_rowPtr[node+1]; ++k) 
    {	
	if (state[sW_colInd[k]])
		value += sW_nnz[k];
       	else
     		value -= sW_nnz[k];
    }

    bool update = value > thresholds[node];
    if (update != state[node]) {
      *stable = false;
      state[node] = update;
    }
  }
  
}

GPUSparseGpuPreProHopfieldNetwork::GPUSparseGpuPreProHopfieldNetwork(const std::vector<float> &thresholds,
                                                   const std::vector<std::vector<float>> &weights,
                                                   float weightThreshold) :
  SparseHopfieldNetwork(thresholds, weights, weightThreshold) {

  //   Convering dense   //
  //   weight matrix to  //
  //    Sparse matrix    //
  hipsparseHandle_t handle;
  hipsparseCreate(&handle);
  int w_size = (int)size;
  float *h_w_dense = (float*)malloc(w_size*w_size*sizeof(*h_w_dense));
#pragma omp parallel for
  for (size_t i = 0; i < size; ++i) {
    for (size_t j = 0; j < size; ++j) {
      //Make loose connections -> No connection
      h_w_dense[i+j*size] = weights[i][j]*weights[i][j]>weightThreshold*weightThreshold ? weights[i][j] : 0;
    }
  }

  
  gpuErrchk(hipMalloc(&d_w_dense,w_size*w_size*sizeof(float)));
  gpuErrchk(hipMemcpy(d_w_dense,h_w_dense,w_size*w_size*sizeof(float),hipMemcpyHostToDevice));

  hipsparseMatDescr_t descrW;
  hipsparseCreateMatDescr(&descrW);
  hipsparseSetMatType (descrW, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase (descrW, HIPSPARSE_INDEX_BASE_ZERO);
  int nnz = 0;
  const int lda = w_size;
  
  gpuErrchk(hipMalloc(&d_nnzPerVector, w_size*sizeof(int)));

  hipsparseSnnz(handle,HIPSPARSE_DIRECTION_ROW,w_size,w_size,descrW,d_w_dense,lda,d_nnzPerVector,&nnz);

  int *h_nnzPerVector = (int *) malloc(w_size*sizeof(int));
  gpuErrchk(hipMemcpy(h_nnzPerVector,d_nnzPerVector,w_size*sizeof(int),hipMemcpyDeviceToHost));


  printf("Percentage of NNZ elements in weight matrix using threshold %f = %f%%\n", weightThreshold,(100.00*nnz/(w_size*w_size)));
 

  //Allocating device memory
  gpuErrchk(hipMalloc((void**)&state_d,sizeof(bool) * size));
  gpuErrchk(hipMalloc((void**)&stable_d,sizeof(bool)));
  gpuErrchk(hipMalloc((void**)&threshold_d,sizeof(float) * size));
  gpuErrchk(hipMalloc((void**)&sW_nnz_d,sizeof(float) * nnz));
  gpuErrchk(hipMalloc((void**)&sW_colInd_d,sizeof(int) * nnz));
  gpuErrchk(hipMalloc((void**)&sW_rowPtr_d,sizeof(int) * (w_size+1)));
  
  hipsparseSdense2csr(handle,w_size,w_size,descrW,d_w_dense,lda,d_nnzPerVector,sW_nnz_d,sW_rowPtr_d,sW_colInd_d);

  // Copying data to device
  gpuErrchk(hipMemcpy(threshold_d, thresholds.data(), size * sizeof(float),hipMemcpyHostToDevice));


}

GPUSparseGpuPreProHopfieldNetwork::~GPUSparseGpuPreProHopfieldNetwork() {

  //Free Device memory
  hipFree(state_d);
  hipFree(threshold_d);
  hipFree(sW_nnz_d);
  hipFree(sW_colInd_d);
  hipFree(sW_rowPtr_d);
  hipFree(stable_d);

}

vector<bool> GPUSparseGpuPreProHopfieldNetwork::evaluate(const vector<bool> &data) {
  // TODO: Implement me!

  bool stable_h;
  bool data_h[size];

  unsigned numThreads = 256;
  unsigned numBlocks = (size-1)/numThreads+1;
  copy(data.begin(), data.end(), data_h);
  gpuErrchk(hipMemcpy(state_d, data_h, size * sizeof(bool),hipMemcpyHostToDevice));
  do {
    stable_h = true;
    gpuErrchk(hipMemcpy(stable_d, &stable_h, sizeof(bool),
                         hipMemcpyHostToDevice));

    gpu_sparse_gpu_prePro_recall_kernel<<< numBlocks, numThreads >>> 
    (size, state_d, threshold_d, sW_nnz_d, sW_colInd_d, sW_rowPtr_d, stable_d);


    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(&stable_h, stable_d, sizeof(bool),
                         hipMemcpyDeviceToHost));
  } while (!stable_h);

  gpuErrchk(hipMemcpy(data_h, state_d, size * sizeof(bool),
                       hipMemcpyDeviceToHost));

  gpuErrchk(hipDeviceSynchronize());
  
  vector<bool> state(data_h, data_h + size);


  return state;

}
