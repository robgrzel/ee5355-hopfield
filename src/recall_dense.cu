#include "hip/hip_runtime.h"

#include "hopfield.hpp"

#include <cstdint>
#include <cassert>
#include <vector>
#include <iostream>
using namespace std;

__global__ void gpu_dense_recall_kernel(size_t size,
                                        bool * state,
                                        float * thresholds,
                                        float * weights,
                                        bool * stable) {
  size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  float value = 0.0f;
  bool update;

  bool stableT = true;

  if (i < size) {
    for (size_t k = 0; k < size; ++k) {
      if (state[k])
        value += weights[i * size + k];
      else
        value -= weights[i * size + k];
    }

    update = value > thresholds[i];
    stableT &= update == state[i];

    state[i] = update;

    //TODO: use reduction to find stable
    atomicAnd((int *) stable, (int) stableT);
  }
}

vector<bool> GPUDenseRecall::recall(const vector<bool> &data,
                                    const vector<float> &thresholds,
                                    const vector<vector<float> > &weights) {
  size_t size = data.size();
  bool stable;

  bool dataArray[size];
  float thresholdArray[size];
  float (*weightArray)[size] = (float(*)[size])new float[size * size];

  bool * stateDev;
  float * thresholdDev;
  float * weightDev;
  bool * stableDev;
  unsigned numThreads = 256;
  unsigned numBlocks = size / numThreads;

  if (size % numThreads) numBlocks++;

  for (size_t i = 0; i < size; ++i) {
    dataArray[i] = data[i];
    thresholdArray[i] = thresholds[i];

    for (size_t j = 0; j < size; ++j) {
      weightArray[i][j] = weights[i][j];
    }
  }

  assert(hipMalloc((void**) &stateDev, sizeof(bool) * size) == hipSuccess);
  assert(hipMalloc((void**) &thresholdDev, sizeof(float) * size)
         == hipSuccess);
  assert(hipMalloc((void**) &weightDev, sizeof(float) * size * size)
         == hipSuccess);
  assert(hipMalloc((void**) &stableDev, sizeof(bool)) == hipSuccess);

  assert(hipMemcpy(stateDev, dataArray, size * sizeof(bool),
                    hipMemcpyHostToDevice) == hipSuccess);
  assert(hipMemcpy(thresholdDev, thresholdArray, size * sizeof(float),
                    hipMemcpyHostToDevice) == hipSuccess);
  assert(hipMemcpy(weightDev, weightArray, size * size * sizeof(float),
                    hipMemcpyHostToDevice) == hipSuccess);

  do {
    stable = true;
    assert(hipMemcpy(stableDev, &stable, sizeof(bool),
                      hipMemcpyHostToDevice) == hipSuccess);

    gpu_dense_recall_kernel<<< numBlocks, numThreads >>>
      (size, stateDev, thresholdDev, weightDev, stableDev);
    assert(hipDeviceSynchronize() == hipSuccess);

    assert(hipMemcpy(&stable, stableDev, sizeof(bool),
                      hipMemcpyDeviceToHost) == hipSuccess);
  } while (!stable);

  assert(hipMemcpy(dataArray, stateDev, size * sizeof(bool),
                    hipMemcpyDeviceToHost) == hipSuccess);

  assert(hipDeviceSynchronize() == hipSuccess);
  
  vector<bool> state(size);
  for (size_t i = 0; i < size; ++i) {
    state[i] = dataArray[i];
  }

  delete[] weightArray;

  hipFree(stateDev);
  hipFree(thresholdDev);
  hipFree(weightDev);
  hipFree(stableDev);

  return state;
}

